﻿#include <complex>

#define M_PI 3.14159265

#include "hip/hip_runtime.h"

#include "hip/hip_complex.h"

__device__ __forceinline__ hipComplex cuComplexExp(hipComplex z)
{
    hipComplex res;
    float t = expf(z.x);
    sincosf(z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

// Kernel definition
__global__ void calculateDFTCUDAKernel(hipComplex* in, hipComplex* out, size_t num)
{
    int i = threadIdx.x;
    if (i < num)
    {
        out[i].x = 0;
        out[i].y = 0;
        for (int j = 0; j < num; j++)
        {
            out[i] = hipCaddf(out[i], 
                hipCmulf(in[j], cuComplexExp(make_hipComplex(0, -2 * M_PI * i * j / num)))
            );
        }
    }
}

void calculateDFTCUDA(std::complex<float>* in, std::complex<float>* out, size_t num)
{
    if (num == 0) return;

    // Allocate vectors in device memory
    hipComplex* d_in;
    hipMalloc(&d_in, num * sizeof(hipComplex));
    hipComplex* d_out;
    hipMalloc(&d_out, num * sizeof(hipComplex));

    // Copy vectors from host memory to device memory
    hipMemcpy(d_in, in, num * sizeof(hipComplex), hipMemcpyHostToDevice);

    calculateDFTCUDAKernel<<<1, num>>>(d_in, d_out, num);
    hipMemcpy(out, d_out, num * sizeof(hipComplex), hipMemcpyDeviceToHost);

    hipError_t cudaStatus;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DFT Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching DFT Kernel!\n", cudaStatus);
        goto Error;
    }

Error:
    hipFree(d_in);
    hipFree(d_out);
}
