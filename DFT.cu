﻿#include <complex>

#define M_PI 3.14159265

#include "hip/hip_runtime.h"

#include "hip/hip_complex.h"

__device__ __forceinline__ hipComplex cuComplexExp(hipComplex z)
{
    hipComplex res;
    float t = expf(z.x);
    sincosf(z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

// Kernel definition
__global__ void calculateDFTCUDAKernel(hipComplex* in, hipComplex* out, size_t num)
{
    int i = threadIdx.x;
    if (i < num)
    {
        out[i].x = 0;
        out[i].y = 0;
        for (int j = 0; j < num; j++)
        {
            out[i] = hipCaddf(out[i], 
                hipCmulf(in[j], cuComplexExp(make_hipComplex(0, -2 * M_PI * i * j / num)))
            );
        }
    }
}

void calculateDFTCUDA(std::complex<float>* in, std::complex<float>* out, size_t num)
{
    if (num == 0) return;

    // Allocate vectors in device memory
    hipComplex* d_in;
    hipMalloc(&d_in, num * sizeof(hipComplex));
    hipComplex* d_out;
    hipMalloc(&d_out, num * sizeof(hipComplex));

    // Copy vectors from host memory to device memory
    hipMemcpy(d_in, in, num * sizeof(hipComplex), hipMemcpyHostToDevice);

    calculateDFTCUDAKernel<<<1, num>>>(d_in, d_out, num);
    hipMemcpy(out, d_out, num * sizeof(hipComplex), hipMemcpyDeviceToHost);

    hipError_t cudaStatus;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DFT Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching DFT Kernel!\n", cudaStatus);
        goto Error;
    }

Error:
    hipFree(d_in);
    hipFree(d_out);
}

// Kernel definition
__global__ void calculateDFTCUDAKernelWithPrecomputedRoot(hipComplex* in, hipComplex* roots, hipComplex* out, size_t num)
{
    int i = threadIdx.x;
    if (i < num)
    {
        out[i].x = 0;
        out[i].y = 0;
        for (int j = 0; j < num; j++)
        {
            out[i] = hipCaddf(out[i], hipCmulf(in[j], roots[i * j % num]));
        }
    }
}

void calculateDFTCUDALargeMem(std::complex<float>* in, std::complex<float>* out, size_t num)
{
    if (num == 0) return;

    // Allocate vectors in device memory
    hipComplex* d_in;
    hipMalloc(&d_in, num * sizeof(hipComplex));
    hipComplex* d_out;
    hipMalloc(&d_out, num * sizeof(hipComplex));
    hipComplex* d_roots;
    hipMalloc(&d_roots, num * sizeof(hipComplex));

    // Copy vectors from host memory to device memory
    hipMemcpy(d_in, in, num * sizeof(hipComplex), hipMemcpyHostToDevice);

    // Precompute DFT
    hipComplex* roots = (hipComplex *)malloc(sizeof(hipComplex) * num);
    for (int i = 0; i < num; i++)
    {
        std::complex<float> root = std::exp(std::complex<float>(0, -2 * M_PI * i / num));
        roots[i] = make_hipComplex(root.real(), root.imag());
    }
    hipMemcpy(d_roots, roots, num * sizeof(hipComplex), hipMemcpyHostToDevice);
    free(roots);

    calculateDFTCUDAKernelWithPrecomputedRoot<<<1, num >>>(d_in, d_roots, d_out, num);

    hipError_t cudaStatus;

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "DFT Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching DFT Kernel!\n", cudaStatus);
        goto Error;
    }

    // Copy back the results
    hipMemcpy(out, d_out, num * sizeof(hipComplex), hipMemcpyDeviceToHost);

Error:
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_roots);
}
